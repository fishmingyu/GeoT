#include "hip/hip_runtime.h"
#include "../../csrc/cuda/index_scatter_kernel.cuh"
#include "./dataloader/dataloader.hpp"
#include "./util/check.cuh"
#include "./util/gpuTimer.cuh"
#include "./util/ramArray.cuh"

#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <fstream>
#include <stdio.h>  // printf
#include <stdlib.h> // EXIT_FAILURE

__global__ void warm_up() {}

// policy listed in template
template <typename ValueType, int NPerThread, int NThreadX, int NnzPerThread,
          int NnzThreadY>
void segscan_sr_sorted(int nnz, int N, util::RamArray<Index> &index,
                       util::RamArray<DType> &src, util::RamArray<DType> &dst) {
  // restriction
  int blockDimX = NThreadX;
  int blockDimY = NnzThreadY;

  dim3 gridDim(CEIL(N, NThreadX * NPerThread),
               CEIL(nnz, NnzThreadY * NnzPerThread), 1);
  dim3 blockDim(blockDimX, blockDimY, 1);

  segscan_sr_sorted_kernel<ValueType, NPerThread, NThreadX, NnzPerThread,
                           NnzThreadY><<<gridDim, blockDim>>>(
      nnz, N, src.d_array.get(), index.d_array.get(), dst.d_array.get());
}

template <typename ValueType, int NPerThread, int NThreadY, int NnzPerThread,
          int RNum, int RSync>
void segscan_pr_sorted(int nnz, int N, util::RamArray<Index> &index,
                       util::RamArray<DType> &src, util::RamArray<DType> &dst) {
  int blockDimX = RSync * RNum;
  int blockDimY = NThreadY;

  dim3 gridDim(CEIL(nnz, RSync * RNum * NnzPerThread),
               CEIL(N, NThreadY * NPerThread), 1);
  dim3 blockDim(blockDimX, blockDimY, 1);

  segscan_pr_sorted_kernel<ValueType, NPerThread, NThreadY, NnzPerThread, RNum,
                           RSync><<<gridDim, blockDim>>>(
      nnz, N, src.d_array.get(), index.d_array.get(), dst.d_array.get());
}

template <typename ValueType>
void check(int nnz, int N, int keys, util::RamArray<int64_t> &index,
           util::RamArray<ValueType> &src, util::RamArray<ValueType> &dst) {
  dst.tocpu();
  src.tocpu();
  index.tocpu();
  util::checkSegScan<ValueType, int64_t>(dst.h_array.get(), src.h_array.get(),
                                         index.h_array.get(), nnz, N, keys);
}

// keys is an estimated value since index may not be continuous
void segscan_sorted(int nnz, int N, int keys, util::RamArray<Index> &index,
                    util::RamArray<DType> &src, util::RamArray<DType> &dst) {
  // restriction
  if (N >= 1 && N <= 4) {
    segscan_pr_sorted<DType, 1, 1, 2, 4, 32>(nnz, N, index, src, dst);
  } else if (N > 4 && N < 32) {
    segscan_pr_sorted<DType, 2, 2, 2, 4, 32>(nnz, N, index, src, dst);
  } else if (N >= 32 && N < 64) {
    int avg_key_len = nnz / keys;
    if (avg_key_len < 16) {
      segscan_sr_sorted<DType, 2, 16, 32, 1>(nnz, N, index, src, dst);
    } else if (avg_key_len >= 16 && avg_key_len < 64) {
      segscan_sr_sorted<DType, 2, 16, 32, 2>(nnz, N, index, src, dst);
    } else {
      segscan_sr_sorted<DType, 2, 16, 32, 4>(nnz, N, index, src, dst);
    }
  } else {
    int avg_key_len = nnz / keys;
    if (avg_key_len < 16) {
      segscan_sr_sorted<DType, 2, 32, 32, 1>(nnz, N, index, src, dst);
    } else if (avg_key_len >= 16 && avg_key_len < 64) {
      segscan_sr_sorted<DType, 2, 32, 32, 2>(nnz, N, index, src, dst);
    } else {
      segscan_sr_sorted<DType, 2, 32, 32, 4>(nnz, N, index, src, dst);
    }
  }
}

int main(int argc, char **argv) {
  // Host problem definition
  if (argc < 3) {
    printf("Input: first get the path of sparse matrix, then get the "
           "feature length of dense matrix\n");
    exit(1);
  }
  char *filename = argv[1];
  int feature_size = atoi(argv[2]);

  const int iter = 300;
  auto indexDescr = DataLoader<DType, Index>(filename);
  int nnz = indexDescr.nnz;
  int keys = indexDescr.keys;

  util::RamArray<DType> src(nnz * feature_size);
  util::RamArray<DType> dst(keys * feature_size);

  src.fill_random_h();
  dst.fill_zero_h();
  // to GPU
  src.tocuda();
  dst.tocuda();
  indexDescr.tocuda();
  printf("start index scatter test\n");
  hipDeviceSynchronize();
  // warm up
  for (int i = 0; i < 1000; i++)
    warm_up<<<1, 1>>>();

  segscan_sorted(nnz, feature_size, keys, indexDescr.sp_indices, src, dst);
  check<DType>(nnz, feature_size, keys, indexDescr.sp_indices, src, dst);
  return 0;
}
