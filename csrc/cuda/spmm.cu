#include "hip/hip_runtime.h"
#ifndef SPMM
#define SPMM

#include "../dataloader/dataloader.hpp"
#include "../util/check.cuh"
#include "../util/ramArray.cuh"
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <iostream>

enum spmm_kernel_met {
  cusparse,
  eb_pr,
  eb_pr_cg,
};

template <typename access_t>
__global__ void csrspmm_parreduce_nnzbalance_kernel(
    const int M, const int N, const int K, const int nnz_,
    const int csr_indptr[], const int csr_indices[], const float csr_data[],
    const float B[], float C[]) {
  constexpr int CoarsenFactor = sizeof(access_t) / sizeof(float);
  int nnz = nnz_;
  if (nnz < 0)
    nnz = csr_indptr[M];

  int lane_id = (threadIdx.x & (32 - 1));
  int Nnzdim_warp_id = blockIdx.x * blockDim.y + threadIdx.y;
  int nz_start = Nnzdim_warp_id * 32;
  int stride = gridDim.x * (blockDim.y * 32);

  // get the dense column offset
  int col_offset = blockIdx.y * 32 + (threadIdx.x >> 5) * CoarsenFactor;
  const float *B_panel = B + col_offset;
  float *C_panel = C + col_offset;
  int ldB = N;
  int ldC = N;

  int k;
  float v;
  float c[CoarsenFactor] = {0};
  float buffer[CoarsenFactor] = {0};

  if (col_offset >= N)
    return;
  if (col_offset + CoarsenFactor >= N)
    goto Ndim_Residue;

  for (int nz_id = nz_start + lane_id;
       nz_id < nnz + lane_id; // make sure NO warp loop-divergence
       nz_id += stride) {
    int row = binary_search_segment_number<int>(csr_indptr, M, nnz, nz_id);

    if (nz_id < nnz) {
      k = csr_indices[nz_id];
      v = __guard_load_default_one<float>(csr_data, nz_id);
    } else {
      k = 0;
      v = 0.0f;
    }

    // load B-elements in vector-type
    *(access_t *)buffer = *(access_t *)(B_panel + k * ldB);
#pragma unroll
    for (int i = 0; i < CoarsenFactor; i++) {
      c[i] = buffer[i] * v;
    }

    // reduction
    int row_intv =
        __shfl_sync(FULLMASK, row, (32 - 1)) - __shfl_sync(FULLMASK, row, 0);
    if (row_intv == 0) {
// if all non-zeros in this warp belong to the same row, use a simple reduction
#pragma unroll
      for (int i = 0; i < CoarsenFactor; i++) {
        SHFL_DOWN_REDUCE(c[i]);
      }
      if (lane_id == 0) {
#pragma unroll
        for (int i = 0; i < CoarsenFactor; i++) {
          atomicAdd(C_panel + row * ldC + i, c[i]);
        }
      }
    } else {
      // if non-zeros belong to different rows, use a parallel-scan primitive
      // thread that holds the start of each segment are responsible for writing
      // results
      bool is_seg_start =
          ((__shfl_up_sync(FULLMASK, row, 1) != row) || (lane_id == 0));
      float tmpv;
      int tmpr;
#pragma unroll
      for (int i = 0; i < CoarsenFactor; i++) {
        SEG_SHFL_SCAN(c[i], tmpv, row, tmpr);
      }
      if (is_seg_start) {
// atomic add has no vector-type form.
#pragma unroll
        for (int i = 0; i < CoarsenFactor; i++) {
          atomicAdd(C_panel + row * ldC + i, c[i]);
        }
      }
    }
  }
  return;
Ndim_Residue:
  int valid_lane_num = N - col_offset;

  for (int nz_id = nz_start + lane_id;
       nz_id < nnz + lane_id; // make sure NO warp loop-divergence
       nz_id += stride) {
    int row = binary_search_segment_number<int>(csr_indptr, M, nnz, nz_id);

    if (nz_id < nnz) {
      k = csr_indices[nz_id];
      v = __guard_load_default_one<float>(csr_data, nz_id);
    } else {
      k = 0;
      v = 0.0f;
    }

#pragma unroll
    for (int i = 0; i < CoarsenFactor; i++) {
      if (i < valid_lane_num) {
        c[i] = B_panel[k * ldB + i] * v;
      }
    }

    // reduction
    int row_intv =
        __shfl_sync(FULLMASK, row, (32 - 1)) - __shfl_sync(FULLMASK, row, 0);
    if (row_intv == 0) {
#pragma unroll
      for (int i = 0; i < CoarsenFactor; i++) {
        SHFL_DOWN_REDUCE(c[i]);
      }
      if (lane_id == 0) {
#pragma unroll
        for (int i = 0; i < CoarsenFactor; i++) {
          if (i < valid_lane_num) {
            atomicAdd(C_panel + row * ldC + i, c[i]);
          }
        }
      }
    } else {
      bool is_seg_start =
          ((__shfl_up_sync(FULLMASK, row, 1) != row) || (lane_id == 0));
      float tmpv;
      int tmpr;
#pragma unroll
      for (int i = 0; i < CoarsenFactor; i++) {
        SEG_SHFL_SCAN(c[i], tmpv, row, tmpr);
      }
      if (is_seg_start) {
#pragma unroll
        for (int i = 0; i < CoarsenFactor; i++) {
          if (i < valid_lane_num) {
            atomicAdd(C_panel + row * ldC + i, c[i]);
          }
        }
      }
    }
  }
  return;
}

template <typename Index, typename DType>
void csrspmm_parreduce_nnzbalance(SpMatCsrDescr_t<Index, DType>& spmatA, 
  const int N, const DType *B, DType *C) {

  // factor of thread coarsening
  int coarsen_factor = (N % 4 == 0) ? 4 : (N % 2 == 0) ? 2 : 1;
  // number of parallel warps along M-dimension
  const int segreduce_size_per_warp = 32;
  int Nnzdim_worker = spmatA.nrow; // CEIL(spmatA.nnz, segreduce_size_per_warp);
  // partition large-N and map to blockdim.y to help cache performance
  int Ndim_threadblock = CEIL(N, 32);
  int Ndim_warp_per_tb = min(N, 32) / coarsen_factor;

  int ref_warp_per_tb = RefThreadPerBlock / 32;
  int Nnzdim_warp_per_tb = CEIL(ref_warp_per_tb, Ndim_warp_per_tb);

  // total number of warps
  int gridDimX = CEIL(Nnzdim_worker, Nnzdim_warp_per_tb);
  int gridDimY = Ndim_threadblock;
  dim3 gridDim(gridDimX, gridDimY, 1);
  dim3 blockDim(Ndim_warp_per_tb * 32, Nnzdim_warp_per_tb, 1);

  if (coarsen_factor == 4) {
  csrspmm_parreduce_nnzbalance_kernel<float4><<<gridDim, blockDim>>>(
  spmatA.nrow, N, spmatA.ncol, spmatA.nnz, spmatA.sp_csrptr.d_array.get(), spmatA.sp_csrind.d_array.get(),
  spmatA.sp_data.d_array.get(), B, C);
  } else if (coarsen_factor == 2) {
  csrspmm_parreduce_nnzbalance_kernel<float2><<<gridDim, blockDim>>>(
  spmatA.nrow, N, spmatA.ncol, spmatA.nnz, spmatA.sp_csrptr.d_array.get(), spmatA.sp_csrind.d_array.get(),
  spmatA.sp_data.d_array.get(), B, C);
  } else {
  csrspmm_parreduce_nnzbalance_kernel<float><<<gridDim, blockDim>>>(
  spmatA.nrow, N, spmatA.ncol, spmatA.nnz, spmatA.sp_csrptr.d_array.get(), spmatA.sp_csrind.d_array.get(),
  spmatA.sp_data.d_array.get(), B, C);
  }
}

template <typename access_t>
__global__ void csrspmm_parreduce_rowbalance_kernel(
    const int M, const int N, const int K, const int csr_indptr[],
    const int csr_indices[], const float csr_data[], const float B[],
    float C[]) {
  constexpr int CoarsenFactor = sizeof(access_t) / sizeof(float);

  int lane_id = (threadIdx.x & (32 - 1));
  int stride = gridDim.x * blockDim.y;
  int row = blockIdx.x * blockDim.y + threadIdx.y;

  // get the dense column offset
  int col_offset = blockIdx.y * 32 + (threadIdx.x >> 5) * CoarsenFactor;
  const float *B_panel = B + col_offset;
  float *C_panel = C + col_offset;
  int ldB = N;
  int ldC = N;

  if (col_offset >= N)
    return;
  if (col_offset + CoarsenFactor >= N)
    goto Ndim_Residue;

  for (; row < M; row += stride) {
    // declare accumulators
    float c[CoarsenFactor] = {0};
    float buffer[CoarsenFactor];

    int start = csr_indptr[row];
    int end = csr_indptr[row + 1];
    int k;
    float v;

    for (int jj = start + lane_id; jj < end; jj += 32) {
      k = csr_indices[jj];
      v = __guard_load_default_one<float>(csr_data, jj);

      // load B-elements in vector-type
      *(access_t *)buffer = *(access_t *)(B_panel + k * ldB);

#pragma unroll
      for (int i = 0; i < CoarsenFactor; i++) {
        c[i] += v * buffer[i];
      }
    }

#pragma unroll
    for (int i = 0; i < CoarsenFactor; i++) {
      // row-wise reduction is a simple merge-tree
      SHFL_DOWN_REDUCE(c[i])
    }

    // store to C in vector-type
    if (lane_id == 0) {
      *(access_t *)(C_panel + row * ldC) = *(access_t *)c;
    }
  }
  return;

Ndim_Residue:
  int valid_lane_num = N - col_offset;

  for (; row < M; row += stride) {
    // get row offsets
    float c[CoarsenFactor] = {0};
    float buffer[CoarsenFactor];
    // access_t res = init_zeros<access_t>();

    int start = csr_indptr[row];
    int end = csr_indptr[row + 1];
    int k;
    float v;

    for (int jj = start + lane_id; jj < end; jj += 32) {
      k = csr_indices[jj];
      v = __guard_load_default_one<float>(csr_data, jj);

#pragma unroll
      for (int i = 0; i < CoarsenFactor; i++) {
        if (i < valid_lane_num) {
          buffer[i] = B_panel[k * ldB + i];
        }
      }

#pragma unroll
      for (int i = 0; i < CoarsenFactor; i++) {
        c[i] += v * buffer[i];
      }
    }

#pragma unroll
    for (int i = 0; i < CoarsenFactor; i++) {
      SHFL_DOWN_REDUCE(c[i])
    }

    if (lane_id == 0) {
#pragma unroll
      for (int i = 0; i < CoarsenFactor; i++) {
        if (i < valid_lane_num) {
          C_panel[row * ldC + i] = c[i];
        }
      }
    }
  }
}

template <typename Index, typename DType>
void csrspmm_parreduce_rowbalance(const SpMatCsrDescr_t<Index, DType>& spmatA, 
  const int N, const DType *B, DType *C) {
  // factor of thread coarsening
  int coarsen_factor = (N % 4 == 0) ? 4 : (N % 2 == 0) ? 2 : 1;
  // number of parallel warps along M-dimension
  int Mdim_worker = spmatA.nrow;
  // partition large-N and map to blockdim.y to help cache performance
  int Ndim_threadblock = CEIL(N, 32);
  int Ndim_warp_per_tb = min(N, 32) / coarsen_factor;

  int ref_warp_per_tb = RefThreadPerBlock / 32;
  int Mdim_warp_per_tb = CEIL(ref_warp_per_tb, Ndim_warp_per_tb);

  // total number of warps
  int gridDimX = CEIL(Mdim_worker, Mdim_warp_per_tb);
  int gridDimY = Ndim_threadblock;
  dim3 gridDim(gridDimX, gridDimY, 1);
  dim3 blockDim(Ndim_warp_per_tb * 32, Mdim_warp_per_tb, 1);

  if (coarsen_factor == 4) {
  csrspmm_parreduce_rowbalance_kernel<float4>
  <<<gridDim, blockDim>>>(spmatA.nrow, N, spmatA.ncol, spmatA.sp_csrptr.d_array.get(),
  spmatA.sp_csrind.d_array.get(), spmatA.sp_data.d_array.get(), B, C);
  } else if (coarsen_factor == 2) {
  csrspmm_parreduce_rowbalance_kernel<float2>
  <<<gridDim, blockDim>>>(spmatA.nrow, N, spmatA.ncol, spmatA.sp_csrptr.d_array.get(),
  spmatA.sp_csrind.d_array.get(), spmatA.sp_data.d_array.get(), B, C);
  } else {
  csrspmm_parreduce_rowbalance_kernel<float>
  <<<gridDim, blockDim>>>(spmatA.nrow, N, spmatA.ncol, spmatA.sp_csrptr.d_array.get(),
  spmatA.sp_csrind.d_array.get(), spmatA.sp_data.d_array.get(), B, C);
  }
}

template <int CoarsenFactor, int ThreadNz>
__global__ void csrspmm_rowcaching_nnzbalance_kernel(
    const int M, const int N, const int K, const int nnz_,
    const int csr_indptr[], const int csr_indices[], const float csr_data[],
    const float B[], float C[]) {
  int nnz = nnz_;
  if (nnz < 0)
    nnz = csr_indptr[M];

  int warp_id = threadIdx.x >> 5;
  int lane_id = threadIdx.x & 31;

  extern __shared__ int shared_mem[];
  int *workspace_rowid = &shared_mem[(warp_id << 5)];
  int *workspace_colid = workspace_rowid + blockDim.x;
  float *workspace_data =
      (float *)(workspace_colid +
                blockDim.x); // float and int has the same size

  // get the sparse-value range of this row
  int global_warp_id = blockIdx.x * (blockDim.x >> 5) + warp_id;
  int nz_start = global_warp_id * (ThreadNz * 32);

  // get the dense column offset
  int col_offset = blockIdx.y * 32 * CoarsenFactor;
  const float *B_lanes[CoarsenFactor];
  float *C_lanes[CoarsenFactor];
#pragma unroll
  for (int i = 0; i < CoarsenFactor; i++) {
    B_lanes[i] = B + col_offset + lane_id + i * 32;
    C_lanes[i] = C + col_offset + lane_id + i * 32;
  }
  int ldB = N;

  // declare accumulators
  float c[CoarsenFactor] = {0.0f};
  int ldC = N;

  int stride = gridDim.x * (blockDim.x >> 5) * ThreadNz * 32;

  if (blockIdx.y == gridDim.y - 1)
    goto Ndim_Residue;

  for (; nz_start < nnz; nz_start += stride) {
  // iterate over the segment of this warp
  for (int tile_base = nz_start; 
    tile_base < min(nz_start + ThreadNz * 32, nnz); tile_base += 32) {

    int thread_nz_id = tile_base + lane_id;
    if (thread_nz_id < nnz) {
      workspace_colid[lane_id] = csr_indices[thread_nz_id];
      workspace_data[lane_id] =
          __guard_load_default_one<float>(csr_data, thread_nz_id);
    } else {
      workspace_colid[lane_id] = 0;
      workspace_data[lane_id] = 0.0f;
    }
    workspace_rowid[lane_id] =
        binary_search_segment_number<int>(csr_indptr, M, nnz, thread_nz_id);
    __syncwarp();

    // initialize with first value
    int k = workspace_colid[0];
    float v = workspace_data[0];
#pragma unroll
    for (int i = 0; i < CoarsenFactor; i++) {
      c[i] = v * B_lanes[i][k * ldB];
    }
    int row_curr = workspace_rowid[0], next_row;

// scan
#pragma unroll
    for (int pp = 1; pp < 32; pp++) {
      next_row = workspace_rowid[pp];
      if (next_row != row_curr) {
#pragma unroll
        for (int i = 0; i < CoarsenFactor; i++) {
          atomicAdd(C_lanes[i] + row_curr * ldC, c[i]);
        }
        row_curr = next_row;
        k = workspace_colid[pp];
        v = workspace_data[pp];
#pragma unroll
        for (int i = 0; i < CoarsenFactor; i++) {
          c[i] = v * B_lanes[i][k * ldB];
        }
      } else {
        k = workspace_colid[pp];
        v = workspace_data[pp];
#pragma unroll
        for (int i = 0; i < CoarsenFactor; i++) {
          c[i] = c[i] + v * B_lanes[i][k * ldB];
        }
      }
    }
#pragma unroll
    for (int i = 0; i < CoarsenFactor; i++) {
      atomicAdd(C_lanes[i] + row_curr * ldC, c[i]);
    }
  }
  }
  return;

Ndim_Residue:

  int valid_lane_num = CEIL(N - col_offset - lane_id, 32);
  
  for (; nz_start < nnz; nz_start += stride) {
  // iterate over the segment of this warp
  for (int tile_base = nz_start; 
    tile_base < min(nz_start + ThreadNz * 32, nnz); tile_base += 32) {

    int thread_nz_id = tile_base + lane_id;
    if (thread_nz_id < nnz) {
      workspace_colid[lane_id] = csr_indices[thread_nz_id];
      workspace_data[lane_id] =
          __guard_load_default_one<float>(csr_data, thread_nz_id);
    } else {
      workspace_colid[lane_id] = 0;
      workspace_data[lane_id] = 0.0f;
    }
    workspace_rowid[lane_id] =
        binary_search_segment_number<int>(csr_indptr, M, nnz, thread_nz_id);
    __syncwarp();

    // initialize with first value
    int k = workspace_colid[0];
    float v = workspace_data[0];
#pragma unroll
    for (int i = 0; i < CoarsenFactor; i++) {
      if (i < valid_lane_num) {
        c[i] = v * B_lanes[i][k * ldB];
      }
    }
    int row_curr = workspace_rowid[0], next_row;

// scan
#pragma unroll
    for (int pp = 1; pp < 32; pp++) {
      next_row = workspace_rowid[pp];
      if (next_row != row_curr) {
#pragma unroll
        for (int i = 0; i < CoarsenFactor; i++) {
          if (i < valid_lane_num) {
            atomicAdd(C_lanes[i] + row_curr * ldC, c[i]);
          }
        }
        row_curr = next_row;
        k = workspace_colid[pp];
        v = workspace_data[pp];
#pragma unroll
        for (int i = 0; i < CoarsenFactor; i++) {
          if (i < valid_lane_num) {
            c[i] = v * B_lanes[i][k * ldB];
          }
        }
      } else {
        k = workspace_colid[pp];
        v = workspace_data[pp];
#pragma unroll
        for (int i = 0; i < CoarsenFactor; i++) {
          if (i < valid_lane_num) {
            c[i] = c[i] + v * B_lanes[i][k * ldB];
          }
        }
      }
    }
#pragma unroll
    for (int i = 0; i < CoarsenFactor; i++) {
      if (i < valid_lane_num) {
        atomicAdd(C_lanes[i] + row_curr * ldC, c[i]);
      }
    }
  }
  }
}

template <typename Index, typename DType>
void csrspmm_rowcaching_nnzbalance(const SpMatCsrDescr_t<Index, DType>& spmatA, 
  const int N, const DType *B, DType *C) {
int coarsen_factor = (N >= 512) ? 4 : (N >= 128) ? 2 : 1;
int Ndim_threadblock = CEIL(N, (32 * coarsen_factor));

// int thread_nz = (spmatA.nnz > 8000 * 128 * 2) ? 2 : 1;
int thread_nz = 1;
int Nnzdim_warp_per_tb = RefThreadPerBlock / 32;
// int Nnzdim_threadblock = CEIL(spmatA.nnz, Nnzdim_warp_per_tb * 32 *
// thread_nz );
int Nnzdim_threadblock = CEIL(spmatA.nrow, Nnzdim_warp_per_tb *thread_nz); // CEIL(spmatA.nnz, Nnzdim_warp_per_tb * 32 * thread_nz );

dim3 gridDim(Nnzdim_threadblock, Ndim_threadblock, 1);
dim3 blockDim(RefThreadPerBlock, 1, 1);

size_t smem_size = (2 * sizeof(int) + sizeof(float)) * RefThreadPerBlock;

// simple heuristic

if (coarsen_factor == 4) {
if (thread_nz == 1)
csrspmm_rowcaching_nnzbalance_kernel<4, 1>
<<<gridDim, blockDim, smem_size>>>(spmatA.nrow, N, spmatA.ncol,
            spmatA.nnz, spmatA.sp_csrptr.d_array.get(), spmatA.sp_csrind.d_array.get(),
            spmatA.sp_data.d_array.get(), B, C);
if (thread_nz == 2)
csrspmm_rowcaching_nnzbalance_kernel<4, 2>
<<<gridDim, blockDim, smem_size>>>(spmatA.nrow, N, spmatA.ncol,
            spmatA.nnz, spmatA.sp_csrptr.d_array.get(), spmatA.sp_csrind.d_array.get(),
            spmatA.sp_data.d_array.get(), B, C);
if (thread_nz == 4)
csrspmm_rowcaching_nnzbalance_kernel<4, 4>
<<<gridDim, blockDim, smem_size>>>(spmatA.nrow, N, spmatA.ncol,
            spmatA.nnz, spmatA.sp_csrptr.d_array.get(), spmatA.sp_csrind.d_array.get(),
            spmatA.sp_data.d_array.get(), B, C);
} else if (coarsen_factor == 2) {
if (thread_nz == 1)
csrspmm_rowcaching_nnzbalance_kernel<2, 1>
<<<gridDim, blockDim, smem_size>>>(spmatA.nrow, N, spmatA.ncol,
            spmatA.nnz, spmatA.sp_csrptr.d_array.get(), spmatA.sp_csrind.d_array.get(),
            spmatA.sp_data.d_array.get(), B, C);
if (thread_nz == 2)
csrspmm_rowcaching_nnzbalance_kernel<2, 2>
<<<gridDim, blockDim, smem_size>>>(spmatA.nrow, N, spmatA.ncol,
            spmatA.nnz, spmatA.sp_csrptr.d_array.get(), spmatA.sp_csrind.d_array.get(),
            spmatA.sp_data.d_array.get(), B, C);
if (thread_nz == 4)
csrspmm_rowcaching_nnzbalance_kernel<2, 4>
<<<gridDim, blockDim, smem_size>>>(spmatA.nrow, N, spmatA.ncol,
            spmatA.nnz, spmatA.sp_csrptr.d_array.get(), spmatA.sp_csrind.d_array.get(),
            spmatA.sp_data.d_array.get(), B, C);
} else {
if (thread_nz == 1)
csrspmm_rowcaching_nnzbalance_kernel<1, 1>
<<<gridDim, blockDim, smem_size>>>(spmatA.nrow, N, spmatA.ncol,
            spmatA.nnz, spmatA.sp_csrptr.d_array.get(), spmatA.sp_csrind.d_array.get(),
            spmatA.sp_data.d_array.get(), B, C);
if (thread_nz == 2)
csrspmm_rowcaching_nnzbalance_kernel<1, 2>
<<<gridDim, blockDim, smem_size>>>(spmatA.nrow, N, spmatA.ncol,
            spmatA.nnz, spmatA.sp_csrptr.d_array.get(), spmatA.sp_csrind.d_array.get(),
            spmatA.sp_data.d_array.get(), B, C);
if (thread_nz == 4)
csrspmm_rowcaching_nnzbalance_kernel<1, 4>
<<<gridDim, blockDim, smem_size>>>(spmatA.nrow, N, spmatA.ncol,
            spmatA.nnz, spmatA.sp_csrptr.d_array.get(), spmatA.sp_csrind.d_array.get(),
            spmatA.sp_data.d_array.get(), B, C);
}
}

__global__ void
csrspmm_seqreduce_rowbalance_kernel(const int nr, const int nv, const int nc,
                                    const int rowPtr[], const int colIdx[],
                                    const float values[], const float dnInput[],
                                    float dnOutput[]) {
  int row_tile = blockDim.y;
  int subwarp_id = threadIdx.y;
  int stride = row_tile * gridDim.x;
  int row = blockIdx.x * row_tile + subwarp_id;
  int v_id = (blockIdx.y * blockDim.x) + threadIdx.x;
  dnInput += v_id;
  dnOutput += v_id;

  float res = 0, val;
  int col;
  for (; row < nr; row += stride) {

    int start = __ldg(rowPtr + row);
    int end = __ldg(rowPtr + row + 1);
    for (int p = start; p < end; p++) {
      col = __ldg(colIdx + p);
      val = __guard_load_default_one<float>(values, p);
      res += val * __ldg(dnInput + col * nv);
    }
    dnOutput[row * nv] = res;
  }
}

template <typename Index, typename DType>
void csrspmm_seqreduce_rowbalance(const SpMatCsrDescr_t<Index, DType>& spmatA, 
  const int N, const DType *B, DType *C) {
  int Mdim_worker = spmatA.nrow;
  int Ndim_worker = N;
  int Ndim_threadblock = CEIL(Ndim_worker, RefThreadPerBlock);
  int Ndim_thread_per_tb = min(Ndim_worker, RefThreadPerBlock);
  int Mdim_thread_per_tb = CEIL(RefThreadPerBlock, Ndim_thread_per_tb);
  int Mdim_threadblock = CEIL(Mdim_worker, Mdim_thread_per_tb);

  dim3 gridDim(Mdim_threadblock, Ndim_threadblock, 1);
  dim3 blockDim(Ndim_thread_per_tb, Mdim_thread_per_tb, 1);

  csrspmm_seqreduce_rowbalance_kernel<<<gridDim, blockDim>>>(
  spmatA.nrow, N, spmatA.ncol, spmatA.sp_csrptr.d_array.get(), spmatA.sp_csrind.d_array.get(),
  spmatA.sp_data.d_array.get(), B, C);
}


template <typename Index, typename DType>
void csrspmm_cusparse(SpMatCsrDescr_t<Index, DType> &spmatA,
                            const int feature_size, DType *in_feature,
                            DType *out_feature) {
  //
  // Run Cusparse-SpMM and check result
  //
  hipsparseHandle_t handle;
  hipsparseSpMatDescr_t csrDescr;
  hipsparseDnMatDescr_t dnMatInputDescr, dnMatOutputDescr;
  float alpha = 1.0f, beta = 0.0f;

  checkCuSparseError(hipsparseCreate(&handle));

  // creating sparse csr matrix
  checkCuSparseError(hipsparseCreateCsr(
      &csrDescr, spmatA.nrow, spmatA.ncol, spmatA.nnz,
      spmatA.sp_csrptr.d_array.get(), spmatA.sp_csrind.d_array.get(),
      spmatA.sp_data.d_array.get(),
      HIPSPARSE_INDEX_32I, // index 32-integer for indptr
      HIPSPARSE_INDEX_32I, // index 32-integer for indices
      HIPSPARSE_INDEX_BASE_ZERO,
      HIP_R_32F // datatype: 32-bit float real number
      ));

  // creating dense matrices
  checkCuSparseError(hipsparseCreateDnMat(&dnMatInputDescr, spmatA.ncol,
                                         feature_size, feature_size, in_feature,
                                         HIP_R_32F, HIPSPARSE_ORDER_ROW));
  checkCuSparseError(hipsparseCreateDnMat(
      &dnMatOutputDescr, spmatA.nrow, feature_size, feature_size, out_feature,
      HIP_R_32F, HIPSPARSE_ORDER_ROW));

  // allocate workspace buffer
  size_t workspace_size;
  checkCuSparseError(hipsparseSpMM_bufferSize(
      handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
      HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, csrDescr, dnMatInputDescr,
      &beta, dnMatOutputDescr, HIP_R_32F, HIPSPARSE_SPMM_ALG_DEFAULT,
      &workspace_size));

  void *workspace = NULL;
  checkCudaError(hipMalloc(&workspace, workspace_size));
  hipDeviceSynchronize();
  gpuErrchk(hipGetLastError());
  // run SpMM
  checkCuSparseError(hipsparseSpMM(handle,
                                  HIPSPARSE_OPERATION_NON_TRANSPOSE, // opA
                                  HIPSPARSE_OPERATION_NON_TRANSPOSE, // opB
                                  &alpha, csrDescr, dnMatInputDescr, &beta,
                                  dnMatOutputDescr, HIP_R_32F,
                                  HIPSPARSE_SPMM_ALG_DEFAULT, workspace));
  hipDeviceSynchronize();
  gpuErrchk(hipGetLastError());
}

template <class Index, class DType, spmm_kernel_met km>
void SpMM_check(SpMatCsrDescr_t<Index, DType>& H, const int feature_size,
util::RamArray<DType> &in_feature, util::RamArray<DType> &out_feature, util::RamArray<DType> &out_ref) {
  out_feature.reset();
  if (km == spmm_kernel_met::cusparse) {
    std::cout<<"cusparse: ";
    csrspmm_cusparse<Index, DType>(H, feature_size, in_feature.d_array.get(),out_feature.d_array.get());
  } else if (km == spmm_kernel_met::eb_pr) {
    std::cout<<"eb_pr: ";
    csrspmm_parreduce_nnzbalance<Index, DType>(H, feature_size, in_feature.d_array.get(),out_feature.d_array.get());
  } else {
    std::cout<<"Not implemented yet!"<<std::endl;
  }
  out_feature.download();
  bool pass = util::check_result(H.nrow, feature_size, out_feature.h_array.get(), out_ref.h_array.get());
  if (pass) {
    std::cout<<"Passed!"<<std::endl;
  } else {
    std::cout<<"Not Passed!"<<std::endl;
}
}

#endif